#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C" {
#include "hostFE.h"
}

#define NUM_THREADS 25
#define NUM_STREAMS 4

__global__ void convKernel(int filter_width,
                           float *filter,
                           int offset,
                           int image_height,
                           int image_width,
                           float *input_image,
                           float *output_image) {
    int2 coord = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y + offset);

    int half_filter_size = filter_width / 2;
    float sum = 0.0f;
    int row, col;
    for (row = -half_filter_size; row <= half_filter_size; row++) {
        for (col = -half_filter_size; col <= half_filter_size; col++) {
            if (coord.y + row >= 0 && coord.y + row < image_height &&
                coord.x + col >= 0 && coord.x + col < image_width) {
                sum += input_image[(coord.y + row) * image_width + coord.x + col] *
                       filter[(row + half_filter_size) * filter_width + col + half_filter_size];
            }
        }
    }
    output_image[coord.y * image_width + coord.x] = sum;
}

extern "C"
void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage, cl_device_id *device,
            cl_context *context, cl_program *program) {
    // Allocate device memory
    int filter_size = filterWidth * filterWidth * sizeof(float);
    int image_size = imageWidth * imageHeight * sizeof(float);
    float *d_filter, *d_input_image, *d_output_image;
    hipMalloc(&d_filter, filter_size);
    hipMalloc(&d_input_image, image_size);
    hipMalloc(&d_output_image, image_size);

    // Pin outputImage
    hipHostRegister(outputImage, image_size, hipHostRegisterPortable);

    // Copy filter and input_image from host to device
    hipMemcpy(d_filter, filter, filter_size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_image, inputImage, image_size, hipMemcpyHostToDevice);

    // Setup streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamCreate(&streams[i]);

    int grid_step = imageHeight / NUM_STREAMS;
    int grid_size = image_size / NUM_STREAMS;

    dim3 block(NUM_THREADS, NUM_THREADS);
    dim3 grid(imageWidth / NUM_THREADS, grid_step / NUM_THREADS);
    int offset = 0;
    for (int i = 0; i < NUM_STREAMS; i++) {
        convKernel<<<grid, block, 0, streams[i]>>>(filterWidth, d_filter, offset, imageHeight, imageWidth,
                                                   d_input_image, d_output_image);
        hipMemcpyAsync(outputImage + imageWidth * offset, d_output_image + imageWidth * offset, grid_size,
                        hipMemcpyDeviceToHost, streams[i]);
        offset += grid_step;
    }

    // Free memory
    hipDeviceSynchronize();
    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamDestroy(streams[i]);
    hipHostUnregister(outputImage);
    hipFree(d_filter);
    hipFree(d_input_image);
    hipFree(d_output_image);
}
