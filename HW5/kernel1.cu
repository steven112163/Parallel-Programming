#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_X 32
#define BLOCK_Y 30

__global__ void mandelKernel(int *d_data,
                             int width,
                             float stepX, float stepY,
                             float lowerX, float lowerY,
                             int maxIteration) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re;
    float z_im = c_im;

    int i;
    float new_re, new_im;
    for (i = 0; i < maxIteration; ++i) {

        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        new_re = z_re * z_re - z_im * z_im;
        new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    d_data[thisX + thisY * width] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations) {
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);
    int *h_data = (int *) malloc(size);
    int *d_data;
    hipMalloc(&d_data, size);

    dim3 threads_per_block(BLOCK_X, BLOCK_Y);
    dim3 num_of_blocks(resX / threads_per_block.x, resY / threads_per_block.y);
    mandelKernel<<<num_of_blocks, threads_per_block>>>(d_data, resX, stepX, stepY, lowerX, lowerY, maxIterations);

    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    memcpy(img, h_data, size);
    hipFree(d_data);
    free(h_data);
}
