#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define RANGE 8

__global__ void mandelKernel(int *d_data,
                             float stepX, float stepY,
                             float lowerX, float lowerY,
                             int maxIteration,
                             int pitch,
                             int range) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re;
    float z_im = c_im;

    int *row = (int *) ((char *) d_data + thisY * pitch);

    int i;
    for (int idx = 0; idx < range; idx++) {
        for (i = 0; i < maxIteration; ++i) {

            if (z_re * z_re + z_im * z_im > 4.f)
                break;

            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = c_re + new_re;
            z_im = c_im + new_im;
        }

        row[thisX + idx] = i;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations) {
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);
    size_t pitch;
    int *h_data, *d_data;
    hipHostAlloc(&h_data, size, hipHostMallocMapped);
    hipMallocPitch(&d_data, &pitch, resX * sizeof(int), resY);

    dim3 threads_per_block(32, 32);
    dim3 num_of_blocks(resX / threads_per_block.x / RANGE, resY / threads_per_block.y);
    mandelKernel<<<num_of_blocks, threads_per_block>>>(d_data,
                                                       stepX, stepY,
                                                       lowerX, lowerY,
                                                       maxIterations,
                                                       pitch,
                                                       RANGE);

    hipMemcpy2D(h_data, resX * sizeof(int), d_data, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_data, size);
    hipHostFree(h_data);
    hipFree(d_data);
}
