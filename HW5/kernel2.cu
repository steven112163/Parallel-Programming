#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NUM_THREADS 8

__global__ void mandelKernel(int *d_data,
                             float stepX, float stepY,
                             float lowerX, float lowerY,
                             int maxIteration,
                             int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re;
    float z_im = c_im;

    int i;
    float new_re, new_im;
    for (i = 0; i < maxIteration; ++i) {

        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        new_re = z_re * z_re - z_im * z_im;
        new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    int *row = (int *) ((char *) d_data + thisY * pitch);
    row[thisX] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations) {
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);
    size_t pitch;
    int *h_data, *d_data;
    hipHostAlloc(&h_data, size, hipHostMallocMapped);
    hipMallocPitch(&d_data, &pitch, resX * sizeof(int), resY);

    dim3 block(NUM_THREADS, NUM_THREADS);
    dim3 grid(resX / NUM_THREADS, resY / NUM_THREADS);
    mandelKernel<<<grid, block>>>(d_data,
                                  stepX, stepY,
                                  lowerX, lowerY,
                                  maxIterations,
                                  pitch);

    hipMemcpy2D(h_data, resX * sizeof(int), d_data, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_data, size);
    hipHostFree(h_data);
    hipFree(d_data);
}
